#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <map>

#include <cusparseLt.h>

#define CHECK_CUDA(func)                                         \
  {                                                              \
    hipError_t status = (func);                                 \
    if (status != hipSuccess)                                   \
    {                                                            \
      printf("CUDA API failed at line %d with error: %s (%d)\n", \
             __LINE__, hipGetErrorString(status), status);      \
    }                                                            \
  }

#define CHECK_CUSPARSE(func)                                         \
  {                                                                  \
    hipsparseStatus_t status = (func);                                \
    if (status != HIPSPARSE_STATUS_SUCCESS)                           \
    {                                                                \
      printf("CUSPARSE API failed at line %d with error: %s (%d)\n", \
             __LINE__, hipsparseGetErrorString(status), status);      \
    }                                                                \
  }

struct Context
{
  cusparseLtHandle_t cslt_handle;
  Context()
  {
    cusparseLtInit(&cslt_handle);
  }
};

extern "C" Context *get_context()
{
  return new Context();
}

void printMatrix(void *A, const char *prefix)
{
  __half hA[16];
  hipMemcpy(hA, A, sizeof(__half) * 16, hipMemcpyDeviceToHost);
  printf("%s", prefix);
  for (int i = 0; i < 16; i++)
  {
    printf("%.3f ", __half2float(hA[i]));
  }
  printf("\n\n");
}

// Everything assumes contiguous memory with stride == num_cols
struct Descriptor
{
  int num_rows;
  int num_cols;
  hipDataType dtype;

  bool operator<(const Descriptor &other) const
  {
    return num_rows < other.num_rows && num_cols < other.num_cols && (int)dtype < (int)other.dtype;
  }
};

static std::map<Descriptor, cusparseLtMatDescriptor_t> dense_descriptors;
cusparseLtMatDescriptor_t *dense_desc_get_or_init(cusparseLtHandle_t *handle, Descriptor desc)
{
  auto it = dense_descriptors.find(desc);
  if (it != dense_descriptors.end())
    return &dense_descriptors[desc];
  else
  {
    cusparseLtMatDescriptor_t mat;
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(handle, &mat, desc.num_rows, desc.num_cols, desc.num_cols, 16, desc.dtype, HIPSPARSE_ORDER_ROW))
    dense_descriptors[desc] = mat;
    return &dense_descriptors[desc];
  }
}

static std::map<Descriptor, cusparseLtMatDescriptor_t> structured_descriptors;
cusparseLtMatDescriptor_t *structured_desc_get_or_init(cusparseLtHandle_t *handle, Descriptor desc)
{
  auto it = structured_descriptors.find(desc);
  if (it != structured_descriptors.end())
    return &structured_descriptors[desc];
  else
  {
    cusparseLtMatDescriptor_t mat;
    CHECK_CUSPARSE(cusparseLtStructuredDescriptorInit(handle, &mat, desc.num_rows, desc.num_cols, desc.num_cols,
                                                      16, desc.dtype, HIPSPARSE_ORDER_ROW, CUSPARSELT_SPARSITY_50_PERCENT))
    structured_descriptors[desc] = mat;
    return &structured_descriptors[desc];
  }
}

struct MatmulDescriptor
{
  hipsparseOperation_t opA;
  hipsparseOperation_t opB;
  Descriptor descA;
  Descriptor descB;
  Descriptor descC;
  cusparseLtMatDescriptor_t *matA;
  cusparseLtMatDescriptor_t *matB;
  cusparseLtMatDescriptor_t *matC;
  cusparseComputeType compute_type;
  bool operator<(const MatmulDescriptor &other) const {
    return (int)opA < (int)other.opA && (int)opB < (int)other.opB && descA < other.descA &&
      descB < other.descB && descC < other.descC && (int)compute_type < (int)other.compute_type
  }
};
static std::map<MatmulDescriptor, cusparseLtMatmulDescriptor_t> matmul_descriptors;
cusparseLtMatmulDescriptor_t *matmul_desc_get_or_init(cusparseLtHandle_t *handle, MatmulDescriptor desc)
{
  auto it = matmul_descriptors.find(desc);
  if (it != matmul_descriptors.end())
    return &matmul_descriptors[desc];
  else
  {
    cusparseLtMatmulDescriptor_t matmul;
    CHECK_CUSPARSE(cusparseLtMatmulDescriptorInit(handle, &matmul, desc.opA, desc.opB, desc.matA, desc.matB, desc.matC,
                                                  desc.matC, desc.compute_type))
    matmul_descriptors[desc] = matmul;
    return &matmul_descriptors[desc];
  }
}

extern "C" void sparse_matmul(void *context, void *A, void *B, void *C, int num_A_rows, int num_A_cols, int num_B_cols)
{
  cusparseLtHandle_t handle = ((Context *)context)->cslt_handle;

  Descriptor descA{num_A_rows, num_A_cols, HIP_R_16F};
  cusparseLtMatDescriptor_t *matA = structured_desc_get_or_init(&handle, descA);
  Descriptor descB{num_A_cols, num_B_cols, HIP_R_16F};
  cusparseLtMatDescriptor_t *matB = dense_desc_get_or_init(&handle, descB);
  Descriptor descC{num_A_rows, num_B_cols, HIP_R_16F};
  cusparseLtMatDescriptor_t *matC = dense_desc_get_or_init(&handle, descC);

  cusparseLtMatmulDescriptor_t *matmul = matmul_desc_get_or_init(&handle, MatmulDescriptor{
    HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_TRANSPOSE, descA, descB, descC, matA, matB, matC, CUSPARSE_COMPUTE_16F
  });

  CHECK_CUSPARSE(cusparseLtSpMMAPrune(&handle, matmul, A, A, CUSPARSELT_PRUNE_SPMMA_STRIP, nullptr))
  // int d_valid;
  // CHECK_CUDA(hipMalloc((void **)&d_valid, sizeof(d_valid)));
  CHECK_CUSPARSE(cusparseLtSpMMAPruneCheck(&handle, matmul, A, nullptr, nullptr))

  __half *A_compressed;
  size_t compressed_size;
  CHECK_CUSPARSE(cusparseLtSpMMACompressedSize2(&handle, matA, &compressed_size))
  CHECK_CUDA(hipMalloc(&A_compressed, compressed_size))
  CHECK_CUSPARSE(cusparseLtSpMMACompress2(&handle, matA, 1, HIPSPARSE_OPERATION_NON_TRANSPOSE, A, A_compressed, nullptr))

  int alg = 0;
  cusparseLtMatmulPlan_t plan;
  cusparseLtMatmulAlgSelection_t alg_sel;
  CHECK_CUSPARSE(cusparseLtMatmulAlgSelectionInit(&handle, &alg_sel, matmul, CUSPARSELT_MATMUL_ALG_DEFAULT))
  CHECK_CUSPARSE(cusparseLtMatmulAlgSetAttribute(&handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg, sizeof(alg)))
  CHECK_CUSPARSE(cusparseLtMatmulPlanInit(&handle, &plan, matmul, &alg_sel, 0))
  float alpha = 1.0f;
  float beta = 0.0f;
  CHECK_CUSPARSE(cusparseLtMatmul(&handle, &plan, &alpha, A_compressed, B, &beta, C, C, nullptr, nullptr, 0))
  CHECK_CUDA(hipFree(A_compressed))
}